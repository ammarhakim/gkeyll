#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_bin_ops_priv.h>
#include <gkyl_dg_calc_gk_rad_vars.h>
#include <gkyl_dg_calc_gk_rad_vars_priv.h>
#include <gkyl_util.h>
}

__global__ void
gkyl_dg_calc_gk_rad_vars_nu_advance_cu_kernel(struct gkyl_dg_calc_gk_rad_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range phase_range,
  struct gkyl_array* vnu_surf, struct gkyl_array* vnu, 
  struct gkyl_array* vsqnu_surf, struct gkyl_array* vsqnu)
{
  double xc[GKYL_MAX_DIM] = {0.0};
  int idx[GKYL_MAX_DIM];
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < phase_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&phase_range, linc1, idx);
    gkyl_rect_grid_cell_center(&up->phase_grid, idx, xc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);
    long loc_phase = gkyl_range_idx(&phase_range, idx);

    const double *bmag_d = (const double*) gkyl_array_cfetch(up->gk_geom->bmag, loc_conf);

    double* vnu_surf_d = (double*) gkyl_array_fetch(vnu_surf, loc_phase);
    double* vnu_d = (double*) gkyl_array_fetch(vnu, loc_phase);
    double* vsqnu_surf_d = (double*) gkyl_array_fetch(vsqnu_surf, loc_phase);  
    double* vsqnu_d = (double*) gkyl_array_fetch(vsqnu, loc_phase);   

    up->rad_nu_vpar(xc, up->phase_grid.dx, up->charge, up->mass, 
      up->a, up->alpha, up->beta, up->gamma, up->v0, 
      bmag_d, vnu_surf_d, vnu_d);
    up->rad_nu_mu(xc, up->phase_grid.dx, up->charge, up->mass, 
      up->a, up->alpha, up->beta, up->gamma, up->v0, 
      bmag_d, vsqnu_surf_d, vsqnu_d);
  }  
}

// Host-side wrapper for radiation drag coefficient calculation
void 
gkyl_dg_calc_gk_rad_vars_nu_advance_cu(const struct gkyl_dg_calc_gk_rad_vars *up,
  const struct gkyl_range *conf_range, const struct gkyl_range *phase_range, 
  struct gkyl_array* vnu_surf, struct gkyl_array* vnu, 
  struct gkyl_array* vsqnu_surf, struct gkyl_array* vsqnu)
{
  int nblocks = phase_range->nblocks;
  int nthreads = phase_range->nthreads;
  gkyl_dg_calc_gk_rad_vars_nu_advance_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, 
    *conf_range, *phase_range, 
    vnu_surf->on_dev, vnu->on_dev, vsqnu_surf->on_dev, vsqnu->on_dev);
}

__global__ void
gkyl_dg_calc_gk_rad_vars_nI_nu_advance_cu_kernel(struct gkyl_dg_calc_gk_rad_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range phase_range,
  const struct gkyl_array* vnu_surf, const struct gkyl_array* vnu, 
  const struct gkyl_array* vsqnu_surf, const struct gkyl_array* vsqnu, 
  const struct gkyl_array *nI, 
  struct gkyl_array* nvnu_surf, struct gkyl_array* nvnu, 
  struct gkyl_array* nvsqnu_surf, struct gkyl_array* nvsqnu)
{
  double xc[GKYL_MAX_DIM] = {0.0};
  int idx[GKYL_MAX_DIM];
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < phase_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&phase_range, linc1, idx);
    gkyl_rect_grid_cell_center(&up->phase_grid, idx, xc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);
    long loc_phase = gkyl_range_idx(&phase_range, idx);

    const double* vnu_surf_d = (const double*) gkyl_array_cfetch(vnu_surf, loc_phase);
    const double* vnu_d = (const double*) gkyl_array_cfetch(vnu, loc_phase);
    const double* vsqnu_surf_d = (const double*) gkyl_array_cfetch(vsqnu_surf, loc_phase);  
    const double* vsqnu_d = (const double*) gkyl_array_cfetch(vsqnu, loc_phase);   

    const double *nI_d = (const double*) gkyl_array_cfetch(nI, loc_conf);

    double* nvnu_surf_d = (double*) gkyl_array_fetch(nvnu_surf, loc_phase);
    double* nvnu_d = (double*) gkyl_array_fetch(nvnu, loc_phase);
    double* nvsqnu_surf_d = (double*) gkyl_array_fetch(nvsqnu_surf, loc_phase);  
    double* nvsqnu_d = (double*) gkyl_array_fetch(nvsqnu, loc_phase);   

    up->rad_nI_nu(vnu_surf_d, vnu_d, vsqnu_surf_d, vsqnu_d, nI_d, 
      nvnu_surf_d, nvnu_d, nvsqnu_surf_d, nvsqnu_d);
  }  
}

// Host-side wrapper for sum_s n_{i_s} nu_s(v) calculation for a given input n_{i_s} and nu_s(v)
void 
gkyl_dg_calc_gk_rad_vars_nI_nu_advance_cu(const struct gkyl_dg_calc_gk_rad_vars *up,
  const struct gkyl_range *conf_range, const struct gkyl_range *phase_range, 
  const struct gkyl_array* vnu_surf, const struct gkyl_array* vnu, 
  const struct gkyl_array* vsqnu_surf, const struct gkyl_array* vsqnu, 
  const struct gkyl_array *nI, 
  struct gkyl_array* nvnu_surf, struct gkyl_array* nvnu, 
  struct gkyl_array* nvsqnu_surf, struct gkyl_array* nvsqnu)
{
  int nblocks = phase_range->nblocks;
  int nthreads = phase_range->nthreads;
  gkyl_dg_calc_gk_rad_vars_nI_nu_advance_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, 
    *conf_range, *phase_range, 
    vnu_surf->on_dev, vnu->on_dev, vsqnu_surf->on_dev, vsqnu->on_dev, 
    nI->on_dev, 
    nvnu_surf->on_dev, nvnu->on_dev, nvsqnu_surf->on_dev, nvsqnu->on_dev);
}

// CUDA kernel to set device pointers to gyrokinetic radiation vars kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_calc_gk_rad_vars_set_cu_dev_ptrs(struct gkyl_dg_calc_gk_rad_vars *up, 
  int cdim, int vdim, int poly_order)
{
  up->rad_nu_vpar = choose_rad_gyrokinetic_nu_vpar_kern(cdim, vdim, poly_order);
  up->rad_nu_mu = choose_rad_gyrokinetic_nu_mu_kern(cdim, vdim, poly_order);
  up->rad_nI_nu = choose_rad_gyrokinetic_nI_nu_kern(cdim, vdim, poly_order);
}

gkyl_dg_calc_gk_rad_vars*
gkyl_dg_calc_gk_rad_vars_cu_dev_new(const struct gkyl_rect_grid *phase_grid, 
  const struct gkyl_basis *conf_basis, const struct gkyl_basis *phase_basis, 
  double charge, double mass, const struct gk_geometry *gk_geom, 
  double a, double alpha, double beta, double gamma, double v0)
{
  struct gkyl_dg_calc_gk_rad_vars *up = (struct gkyl_dg_calc_gk_rad_vars*) gkyl_malloc(sizeof(*up));

  up->phase_grid = *phase_grid;
  int cdim = conf_basis->ndim;
  int pdim = phase_basis->ndim;
  int vdim = pdim - cdim;
  int poly_order = phase_basis->poly_order;
  up->cdim = cdim;
  up->pdim = pdim;

  up->charge = charge;
  up->mass = mass;

  // Fitting parameters for a given collision type
  up->a = a;
  up->alpha = alpha;
  up->beta = beta;
  up->gamma = gamma;
  up->v0 = v0;

  // acquire pointer to geometry object
  struct gk_geometry *geom = gkyl_gk_geometry_acquire(gk_geom);
  up->gk_geom = geom->on_dev; // this is so the memcpy below has geometry on_dev

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_calc_gk_rad_vars *up_cu = (struct gkyl_dg_calc_gk_rad_vars*) gkyl_cu_malloc(sizeof(*up_cu));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_calc_gk_rad_vars), GKYL_CU_MEMCPY_H2D);

  dg_calc_gk_rad_vars_set_cu_dev_ptrs<<<1,1>>>(up_cu, cdim, vdim, poly_order);

  // set parent on_dev pointer
  up->on_dev = up_cu;

  // updater should store host pointers
  up->gk_geom = geom; 
  
  return up;
}
