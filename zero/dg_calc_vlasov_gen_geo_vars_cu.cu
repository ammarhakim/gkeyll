#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_bin_ops_priv.h>
#include <gkyl_dg_calc_vlasov_gen_geo_vars.h>
#include <gkyl_dg_calc_vlasov_gen_geo_vars_priv.h>
#include <gkyl_util.h>
}

static void
gkyl_parallelize_components_kernel_launch_dims(dim3* dimGrid, dim3* dimBlock, gkyl_range range, int ncomp)
{
  // Create a 2D thread grid so we launch ncomp*range.volume number of threads and can parallelize over components too
  dimBlock->y = ncomp;
  dimGrid->y = 1;
  dimBlock->x = gkyl_int_div_up(252, ncomp);
  dimGrid->x = gkyl_int_div_up(range.volume, dimBlock->x);
}

__global__ void
gkyl_dg_calc_vlasov_gen_geo_vars_alpha_surf_cu_kernel(struct gkyl_dg_calc_vlasov_gen_geo_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range phase_range, struct gkyl_range phase_ext_range, 
  struct gkyl_array* alpha_surf, struct gkyl_array* sgn_alpha_surf, struct gkyl_array* const_sgn_alpha)
{ 
  int pdim = up->pdim;
  int cdim = up->cdim;
  int idx[GKYL_MAX_DIM], idx_edge[GKYL_MAX_DIM];
  double xc[GKYL_MAX_DIM];

  // 2D thread grid
  // linc2 = c where c is the component index (from 0 to cdim + 1)
  long linc2 = threadIdx.y + blockIdx.y*blockDim.y;
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < phase_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&phase_range, linc1, idx);
    gkyl_rect_grid_cell_center(&up->phase_grid, idx, xc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);
    long loc_phase = gkyl_range_idx(&phase_range, idx);

    const double *tvComp_d = (const double*) gkyl_array_cfetch(up->gk_geom->dxdz, loc_conf);
    const double *gij_d = (const double*) gkyl_array_cfetch(up->gk_geom->gij, loc_conf);

    double* alpha_surf_d = (double*) gkyl_array_fetch(alpha_surf, loc_phase);
    double* sgn_alpha_surf_d = (double*) gkyl_array_fetch(sgn_alpha_surf, loc_phase);
    int* const_sgn_alpha_d = (int*) gkyl_array_fetch(const_sgn_alpha, loc_phase);
    for (int dir = 0; dir<cdim+1; ++dir) {
      // Each thread in linc2 thread grid handles a different component
      if (linc2 == dir) {
        const_sgn_alpha_d[dir] = up->alpha_surf[dir](xc, up->phase_grid.dx, 
          tvComp_d, gij_d, alpha_surf_d, sgn_alpha_surf_d);

        // If the phase space index is at the local configuration space upper value, we
        // we are at the configuration space upper edge and we also need to evaluate 
        // alpha = +1 to avoid evaluating the geometry information in the ghost cells 
        // where it is not defined when computing the final surface alpha we need
        // (since the surface alpha array stores only the *lower* surface expansion)
        if (dir < cdim && idx[dir] == conf_range.upper[dir]) {
          gkyl_copy_int_arr(pdim, idx, idx_edge);
          idx_edge[dir] = idx_edge[dir]+1;
          long loc_phase_ext = gkyl_range_idx(&phase_ext_range, idx_edge);

          double* alpha_surf_ext_d = (double*) gkyl_array_fetch(alpha_surf, loc_phase_ext);
          double* sgn_alpha_surf_ext_d = (double*) gkyl_array_fetch(sgn_alpha_surf, loc_phase_ext);
          int* const_sgn_alpha_ext_d = (int*) gkyl_array_fetch(const_sgn_alpha, loc_phase_ext);
          const_sgn_alpha_ext_d[dir] = up->alpha_edge_surf[dir](xc, up->phase_grid.dx, 
            tvComp_d, gij_d, alpha_surf_ext_d, sgn_alpha_surf_ext_d);
        }  
      }
    }
  }
}

// Host-side wrapper for general geometry vlasov surface alpha calculation
void gkyl_dg_calc_vlasov_gen_geo_vars_alpha_surf_cu(struct gkyl_dg_calc_vlasov_gen_geo_vars *up, 
  const struct gkyl_range *conf_range, const struct gkyl_range *phase_range, const struct gkyl_range *phase_ext_range, 
  struct gkyl_array* alpha_surf, struct gkyl_array* sgn_alpha_surf, struct gkyl_array* const_sgn_alpha)
{
  dim3 dimGrid, dimBlock;
  gkyl_parallelize_components_kernel_launch_dims(&dimGrid, &dimBlock, *phase_range, up->cdim);
  gkyl_dg_calc_vlasov_gen_geo_vars_alpha_surf_cu_kernel<<<dimGrid, dimBlock>>>(up->on_dev, 
    *conf_range, *phase_range, *phase_ext_range, 
    alpha_surf->on_dev, sgn_alpha_surf->on_dev, const_sgn_alpha->on_dev);
}

__global__ void
gkyl_dg_calc_vlasov_gen_geo_vars_cot_vec_cu_kernel(struct gkyl_dg_calc_vlasov_gen_geo_vars *up, 
  struct gkyl_range conf_range, struct gkyl_array* cot_vec)
{ 
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);

    const double *tvComp_d = (const double*) gkyl_array_cfetch(up->gk_geom->dxdz, loc_conf);
    const double *gij_d = (const double*) gkyl_array_cfetch(up->gk_geom->gij, loc_conf);

    double* cot_vec_d = (double*) gkyl_array_fetch(cot_vec, loc_conf);
    up->calc_cot_vec(tvComp_d, gij_d, cot_vec_d);
  }
}

// Host-side wrapper for contangent vector calculation 
void
gkyl_dg_calc_vlasov_gen_geo_vars_cot_vec_cu(struct gkyl_dg_calc_vlasov_gen_geo_vars *up, 
  const struct gkyl_range *conf_range, struct gkyl_array* cot_vec)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_vlasov_gen_geo_vars_cot_vec_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, 
    *conf_range, cot_vec->on_dev);
}

// CUDA kernel to set device pointers to pkpm vars kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_calc_vlasov_gen_geo_vars_set_cu_dev_ptrs(struct gkyl_dg_calc_vlasov_gen_geo_vars *up, 
  int cdim, int poly_order)
{
  for (int d=0; d<cdim; ++d) {
    up->alpha_surf[d] = choose_vlasov_gen_geo_alpha_surf_kern(d, cdim, poly_order);
    up->alpha_edge_surf[d] = choose_vlasov_gen_geo_alpha_edge_surf_kern(d, cdim, poly_order);
  }
  up->calc_cot_vec = choose_vlasov_gen_geo_cot_vec_kern(cdim, poly_order);
}

gkyl_dg_calc_vlasov_gen_geo_vars*
gkyl_dg_calc_vlasov_gen_geo_vars_cu_dev_new(const struct gkyl_rect_grid *phase_grid, 
  const struct gkyl_basis *conf_basis, const struct gkyl_basis *phase_basis, 
  const struct gk_geometry *gk_geom)
{
  struct gkyl_dg_calc_vlasov_gen_geo_vars *up = (struct gkyl_dg_calc_vlasov_gen_geo_vars*) gkyl_malloc(sizeof(*up));

  up->phase_grid = *phase_grid;
  int cdim = conf_basis->ndim;
  int pdim = phase_basis->ndim;
  int poly_order = phase_basis->poly_order;
  up->cdim = cdim;
  up->pdim = pdim;

  // acquire pointer to geometry object
  struct gk_geometry *geom = gkyl_gk_geometry_acquire(gk_geom);
  up->gk_geom = geom->on_dev; // this is so the memcpy below has geometry on_dev

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_calc_vlasov_gen_geo_vars *up_cu = (struct gkyl_dg_calc_vlasov_gen_geo_vars*) gkyl_cu_malloc(sizeof(*up_cu));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_calc_vlasov_gen_geo_vars), GKYL_CU_MEMCPY_H2D);

  dg_calc_vlasov_gen_geo_vars_set_cu_dev_ptrs<<<1,1>>>(up_cu, cdim, poly_order);

  // set parent on_dev pointer
  up->on_dev = up_cu;

  // updater should store host pointers
  up->gk_geom = geom; 
  
  return up;
}
