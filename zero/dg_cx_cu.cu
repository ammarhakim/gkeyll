#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array.h>
#include <gkyl_array_ops.h>
#include <gkyl_dg_bin_ops.h>
#include <gkyl_dg_cx.h>
#include <gkyl_dg_cx_priv.h>
#include <gkyl_util.h>
#include <gkyl_const.h>
}

// CUDA kernel to set device pointers to kernels.
__global__ static void
gkyl_dg_cx_set_cu_ker_ptrs(struct gkyl_dg_cx_kernels *kernels,
  struct gkyl_basis pbasis_vl, int tblidx)
{
  enum gkyl_basis_type b_type = pbasis_vl.b_type;
  int poly_order = pbasis_vl.poly_order;

  switch (b_type) {
    case GKYL_BASIS_MODAL_HYBRID:
    case GKYL_BASIS_MODAL_SERENDIPITY:
      kernels->react_rate = ser_cx_react_rate_kernels[tblidx].kernels[poly_order];
      break;
    default:
      assert(false);
      break;
  }
};

void
dg_cx_choose_kernel_cu(struct gkyl_dg_cx_kernels *kernels,
  struct gkyl_basis pbasis_vl, struct gkyl_basis cbasis)
{
  int pdim = pbasis_vl.ndim;
  int cdim = cbasis.ndim;
  int vdim = pdim - cdim;

  assert(cv_index[cdim].vdim[vdim] != -1);
  gkyl_dg_cx_set_cu_ker_ptrs<<<1,1>>>(kernels, pbasis_vl, cv_index[cdim].vdim[vdim]);
}

__global__ static void
gkyl_cx_react_rate_cu_ker(struct gkyl_dg_cx_kernels *kernels, const struct gkyl_range conf_rng, 
  const struct gkyl_array *prim_vars_ion, const struct gkyl_array *prim_vars_neut, const struct gkyl_array *upar_b_i, 
  double vt_sq_ion_min, double vt_sq_neut_min, struct gkyl_array *coef_cx,
  double a, double b)
{
  int cidx[GKYL_MAX_CDIM];
  for(unsigned long tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < conf_rng.volume; tid += blockDim.x*gridDim.x) {
    gkyl_sub_range_inv_idx(&conf_rng, tid, cidx);
    long loc = gkyl_range_idx(&conf_rng, cidx);

    const double *prim_vars_ion_d = (const double*) gkyl_array_cfetch(prim_vars_ion, loc);
    const double *prim_vars_neut_d = (const double*) gkyl_array_cfetch(prim_vars_neut, loc);
    const double *upar_b_i_d = (const double*) gkyl_array_cfetch(upar_b_i, loc);

    double *coef_cx_d = (double*) gkyl_array_fetch(coef_cx, loc);

    // call the cx kernel
    double cflr = kernels->react_rate(a, b, vt_sq_ion_min, vt_sq_neut_min, 
      prim_vars_ion_d, prim_vars_neut_d, upar_b_i_d, coef_cx_d);
  }
}

void gkyl_dg_cx_coll_cu(const struct gkyl_dg_cx *up, 
  struct gkyl_array *prim_vars_ion, struct gkyl_array *prim_vars_neut,
  struct gkyl_array *upar_b_i, struct gkyl_array *coef_cx, struct gkyl_array *cflrate)
{  
  gkyl_cx_react_rate_cu_ker<<<up->conf_rng->nblocks, up->conf_rng->nthreads>>>(up->kernels, *up->conf_rng,
    prim_vars_ion->on_dev, prim_vars_neut->on_dev, upar_b_i->on_dev, 
    up->vt_sq_ion_min, up->vt_sq_neut_min, coef_cx->on_dev, up->a, up->b);
}
