#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <assert.h>
#include <stdlib.h>
#include <string.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_mom_gyrokinetic.h>
#include <gkyl_mom_gyrokinetic_priv.h>
#include <gkyl_util.h>
}

enum { M0, M1, M2, M2par, M2perp, M3par, M3perp, ThreeMoments, FourMoments, BAD };

static int
get_gk_mom_id(const char *mom)
{
  int mom_idx = BAD;

  if (strcmp(mom, "M0") == 0) { // density
    mom_idx = M0;
  }
  else if (strcmp(mom, "M1") == 0) { // parallel momentum
    mom_idx = M1;
  }
  else if (strcmp(mom, "M2") == 0) { // total energy
    mom_idx = M2;
  }
  else if (strcmp(mom, "M2par") == 0) { // parallel energy
    mom_idx = M2par;
  }
  else if (strcmp(mom, "M2perp") == 0) { // perpendicular energy
    mom_idx = M2perp;
  }
  else if (strcmp(mom, "M3par") == 0) { // parallel heat flux
    mom_idx = M3par;
  }
  else if (strcmp(mom, "M3perp") == 0) { // perpendicular heat flux
    mom_idx = M3perp;
  }
  else if (strcmp(mom, "ThreeMoments") == 0) {
    // Zeroth (density), First (parallel momentum), 
    // and Second (total energy) computed together
    mom_idx = ThreeMoments;                    
  }
  else if (strcmp(mom, "FourMoments") == 0) {
    // Density, parallel momentum, parallel and
    // perpendicular kinetic energy.
    mom_idx = FourMoments;
  }
  else {
    mom_idx = BAD;
  }    

  return mom_idx;
}

static int
gk_num_mom(int vdim, int mom_id)
{
  int num_mom = 0;
  
  switch (mom_id) {
    case M0:
      num_mom = 1;
      break;

    case M1:
      num_mom = 1;
      break;

    case M2:
      num_mom = 1;
      break;

    case M2par:
      num_mom = 1;
      break;

    case M2perp:
      num_mom = 1;
      break;

    case M3par:
      num_mom = 1;
      break;

    case M3perp:
      num_mom = 1;
      break;

    case ThreeMoments:
      num_mom = 3;
      break;      
      
    case FourMoments:
      num_mom = vdim>1? 4 : 3;
      break;      
      
    default: // can't happen
      break;
  }

  return num_mom;
}

__global__
static void
set_cu_ptrs(struct mom_type_gyrokinetic *mom_gk,
  int mom_id, enum gkyl_basis_type b_type, int vdim, int poly_order, int tblidx)
{
  // choose kernel tables based on basis-function type
  const gkyl_gyrokinetic_mom_kern_list *m0_kernels, *m1_kernels, *m2_kernels, 
    *m2_par_kernels, *m2_perp_kernels, *m3_par_kernels, *m3_perp_kernels,
    *three_moments_kernels, *four_moments_kernels;
  
  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      m0_kernels = ser_m0_kernels;
      m1_kernels = ser_m1_kernels;
      m2_kernels = ser_m2_kernels;
      m2_par_kernels = ser_m2_par_kernels;
      m2_perp_kernels = ser_m2_perp_kernels;
      m3_par_kernels = ser_m3_par_kernels;
      m3_perp_kernels = ser_m3_perp_kernels;
      three_moments_kernels = ser_three_moments_kernels;
      four_moments_kernels = ser_four_moments_kernels;
      break;

    default:
      assert(false);
      break;    
  }  
  
  switch (mom_id) {
    case M0:
      mom_gk->momt.kernel = m0_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = 1;
      break;

    case M1:
      mom_gk->momt.kernel = m1_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = 1;
      break;

    case M2:
      mom_gk->momt.kernel = m2_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = 1;
      break;

    case M2par:
      mom_gk->momt.kernel = m2_par_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = 1;
      break;

    case M2perp:
      mom_gk->momt.kernel = m2_perp_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = 1;
      break;

    case M3par:
      mom_gk->momt.kernel = m3_par_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = 1;
      break;

    case M3perp:
      mom_gk->momt.kernel = m3_perp_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = 1;
      break;

    case ThreeMoments:
      mom_gk->momt.kernel = three_moments_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = 3;
      break;
      
    case FourMoments:
      mom_gk->momt.kernel = four_moments_kernels[tblidx].kernels[poly_order];
      mom_gk->momt.num_mom = vdim>1? 4 : 3;
      break;
      
    default: // can't happen
      break;
  }
}

struct gkyl_mom_type*
gkyl_mom_gyrokinetic_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_basis* pbasis, 
  const struct gkyl_range* conf_range, double mass, const struct gkyl_velocity_map* vel_map,
  const struct gk_geometry *gk_geom, const char *mom)
{
  assert(cbasis->poly_order == pbasis->poly_order);

  struct mom_type_gyrokinetic *mom_gk = (struct mom_type_gyrokinetic*)
    gkyl_malloc(sizeof(struct mom_type_gyrokinetic));
  
  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  mom_gk->momt.cdim = cdim;
  mom_gk->momt.pdim = pdim;
  mom_gk->momt.poly_order = poly_order;
  mom_gk->momt.num_config = cbasis->num_basis;
  mom_gk->momt.num_phase = pbasis->num_basis;

  int mom_id = get_gk_mom_id(mom);
  if(mom_id == BAD) {
     printf("Error: requested GK moment %s not valid\n", mom);
     assert(mom_id != BAD);
  }
  mom_gk->momt.num_mom = gk_num_mom(vdim, mom_id); // number of moments

  mom_gk->mass = mass;

  // Acquire pointers to on_dev objects so memcpy below copies those too.
  struct gk_geometry *geom_ho = gkyl_gk_geometry_acquire(gk_geom);
  struct gkyl_velocity_map *vel_map_ho = gkyl_velocity_map_acquire(vel_map);
  mom_gk->gk_geom = geom_ho->on_dev;
  mom_gk->vel_map = vel_map_ho->on_dev;

  mom_gk->conf_range = *conf_range;

  mom_gk->momt.flags = 0;
  GKYL_SET_CU_ALLOC(mom_gk->momt.flags);
  mom_gk->momt.ref_count = gkyl_ref_count_init(gkyl_gk_mom_free);
  
  // copy struct to device
  struct mom_type_gyrokinetic *mom_gk_cu = (struct mom_type_gyrokinetic*)
    gkyl_cu_malloc(sizeof(struct mom_type_gyrokinetic));
  gkyl_cu_memcpy(mom_gk_cu, mom_gk, sizeof(struct mom_type_gyrokinetic), GKYL_CU_MEMCPY_H2D);

  assert(cv_index[cdim].vdim[vdim] != -1);

  set_cu_ptrs<<<1,1>>>(mom_gk_cu, mom_id, cbasis->b_type,
    vdim, poly_order, cv_index[cdim].vdim[vdim]);

  mom_gk->momt.on_dev = &mom_gk_cu->momt;

  // Updater should store host pointers.
  mom_gk->gk_geom = geom_ho; 
  mom_gk->vel_map = vel_map_ho; 
  
  return &mom_gk->momt;
}

__global__
static void
set_int_cu_ptrs(struct mom_type_gyrokinetic* momt, enum gkyl_basis_type b_type, int vdim,
  int poly_order, int tblidx)
{
  // set kernel pointer
  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      momt->momt.kernel = ser_int_mom_kernels[tblidx].kernels[poly_order];
      break;

    default:
      assert(false);
      break;    
  }
}

struct gkyl_mom_type*
gkyl_int_mom_gyrokinetic_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_basis* pbasis, 
  const struct gkyl_range* conf_range, double mass, const struct gkyl_velocity_map* vel_map,
  const struct gk_geometry *gk_geom)
{
  assert(cbasis->poly_order == pbasis->poly_order);

  struct mom_type_gyrokinetic *momt = (struct mom_type_gyrokinetic*)
    gkyl_malloc(sizeof(struct mom_type_gyrokinetic));
  
  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  momt->momt.cdim = cdim;
  momt->momt.pdim = pdim;
  momt->momt.poly_order = poly_order;
  momt->momt.num_config = cbasis->num_basis;
  momt->momt.num_phase = pbasis->num_basis;

  momt->momt.num_mom = vdim+2;

  momt->mass = mass;
  // Acquire pointers to on_dev objects so memcpy below copies those too.
  struct gk_geometry *geom_ho = gkyl_gk_geometry_acquire(gk_geom);
  struct gkyl_velocity_map *vel_map_ho = gkyl_velocity_map_acquire(vel_map);
  momt->gk_geom = geom_ho->on_dev;
  momt->vel_map = vel_map_ho->on_dev;

  momt->conf_range = *conf_range;

  momt->momt.flags = 0;
  GKYL_SET_CU_ALLOC(momt->momt.flags);
  momt->momt.ref_count = gkyl_ref_count_init(gkyl_gk_mom_free);
  
  // copy struct to device
  struct mom_type_gyrokinetic *momt_cu = (struct mom_type_gyrokinetic*)
    gkyl_cu_malloc(sizeof(struct mom_type_gyrokinetic));
  gkyl_cu_memcpy(momt_cu, momt, sizeof(struct mom_type_gyrokinetic), GKYL_CU_MEMCPY_H2D);

  set_int_cu_ptrs<<<1,1>>>(momt_cu, cbasis->b_type,
    vdim, poly_order, cv_index[cdim].vdim[vdim]);

  momt->momt.on_dev = &momt_cu->momt;

  // Updater should store host pointers.
  momt->gk_geom = geom_ho; 
  momt->vel_map = vel_map_ho; 
  
  return &momt->momt;
}
